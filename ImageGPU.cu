#include "ImageGPU.h"

#include "hip/hip_runtime_api.h"
#include ""

ImageGPU::ImageGPU(const ImageDTOForGPU& image)
	: pPixel(nullptr)
	, Width(image.width)
	, Height(image.height)
	, FrequencyTable{ 0, }
{
	hipError_t errorCode = hipMalloc(&pPixel, sizeof(Pixel) * Width * Height);
	if (errorCode != hipSuccess)
	{
		printf("%s - %s\n", hipGetErrorName(errorCode), hipGetErrorString(errorCode));
	}
}

ImageGPU::~ImageGPU()
{
	hipFree(pPixel);
}
