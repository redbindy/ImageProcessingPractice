#include "hip/hip_runtime.h"
#include "ProcessingHelperGPU.h"

#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>

__global__ void EqualizeHelperComputePixel(
	Pixel* pPixelsGPU,
	const int imageWidth,
	const int imageHeight,
	FrequencyTable* pFrequencyTableGPU
)
{
	const int pixelCount = imageWidth * imageHeight;

	const unsigned int col = (blockDim.x * blockIdx.x) + threadIdx.x;
	if (col >= imageWidth)
	{
		return;
	}

	const unsigned int row = (blockDim.y * blockIdx.y) + threadIdx.y;
	if (row >= imageHeight)
	{
		return;
	}

	const unsigned int index = row * imageWidth + col;

	__shared__ float cdf[TABLE_SIZE];

	if (threadIdx.x == 0 && threadIdx.y == 0)
	{
		uint32_t sum = 0;
		for (int i = 0; i < TABLE_SIZE; ++i)
		{
			switch (blockIdx.z)
			{
			case EHandleColor::RED:
				{
					sum += pFrequencyTableGPU->redTable[i];
				}
				break;

			case EHandleColor::GREEN:
				{
					sum += pFrequencyTableGPU->greenTable[i];
				}
				break;

			case EHandleColor::BLUE:
				{
					sum += pFrequencyTableGPU->blueTable[i];
				}
				break;

			default:
				printf("Invalid input %d\n", blockIdx.z);
				break;
			}

			cdf[i] = sum / (float)pixelCount;
		}
	}

	__syncthreads();

	Pixel* pPixel = pPixelsGPU + index;
	switch (blockIdx.z)
	{
	case EHandleColor::RED:
		{
			pPixel->rgba.r = (uint8_t)round(UINT8_MAX * cdf[pPixel->rgba.r]);
		}
		break;

	case EHandleColor::GREEN:
		{
			pPixel->rgba.g = (uint8_t)round(UINT8_MAX * cdf[pPixel->rgba.g]);
		}
		break;

	case EHandleColor::BLUE:
		{
			pPixel->rgba.b = (uint8_t)round(UINT8_MAX * cdf[pPixel->rgba.b]);
		}
		break;

	default:
		printf("Invalid input %d\n", blockIdx.z);
		break;
	}
}

__global__ void EqualizeHelperGetFrequencyTable(
	Pixel* pPixelsGPU,
	const int imageWidth,
	const int imageHeight,
	FrequencyTable* pFrequencyTableGPU
)
{
	const int pixelCount = imageWidth * imageHeight;

	const unsigned int col = (blockDim.x * blockIdx.x) + threadIdx.x;
	if (col >= imageWidth)
	{
		return;
	}

	const unsigned int row = (blockDim.y * blockIdx.y) + threadIdx.y;
	if (row >= imageHeight)
	{
		return;
	}

	const unsigned int index = row * imageWidth + col;

	Pixel* pPixel = pPixelsGPU + index;
	switch (blockIdx.z)
	{
	case EHandleColor::RED:
		{
			atomicAdd(pFrequencyTableGPU->redTable + pPixel->rgba.r, 1);
		}
		break;

	case EHandleColor::GREEN:
		{
			atomicAdd(pFrequencyTableGPU->greenTable + pPixel->rgba.g, 1);
		}
		break;

	case EHandleColor::BLUE:
		{
			atomicAdd(pFrequencyTableGPU->blueTable + pPixel->rgba.b, 1);
		}
		break;

	default:
		printf("Invalid input %d\n", blockIdx.z);
		break;
	}
}

void EqualizeHelperGPU(
	Pixel* pPixels,
	const int imageWidth,
	const int imageHeight,
	FrequencyTable* pFrequencyTable
)
{
	const int pixelCount = imageWidth * imageHeight;

	Pixel* pPixelsGPU = nullptr;
	FrequencyTable* pFrequencyTableGPU = nullptr;

	hipError_t errorCode = hipMalloc(&pPixelsGPU, sizeof(Pixel) * pixelCount);
	if (errorCode != hipSuccess)
	{
		printf("%s - %s\n", hipGetErrorName(errorCode), hipGetErrorString(errorCode));
		return;
	}

	errorCode = hipMalloc(&pFrequencyTableGPU, sizeof(FrequencyTable));
	if (errorCode != hipSuccess)
	{
		printf("%s - %s\n", hipGetErrorName(errorCode), hipGetErrorString(errorCode));
		return;
	}

	{
		dim3 blockDim = { 32, 32, 1 };
		dim3 gridDim = {
			(unsigned int)ceil(imageWidth / (float)blockDim.x),
			(unsigned int)ceil(imageHeight / (float)blockDim.y),
			EHandleColor::COUNT
		};

		hipMemcpy(pPixelsGPU, pPixels, pixelCount * sizeof(Pixel), hipMemcpyHostToDevice);
		hipMemcpy(pFrequencyTableGPU, pFrequencyTable, sizeof(FrequencyTable), hipMemcpyHostToDevice);
		{
			EqualizeHelperComputePixel << <gridDim, blockDim >> > (pPixelsGPU, imageWidth, imageHeight, pFrequencyTableGPU);
		}
		hipMemcpy(pPixels, pPixelsGPU, pixelCount * sizeof(Pixel), hipMemcpyDeviceToHost);
		hipMemset(pFrequencyTableGPU, 0, sizeof(FrequencyTable));
		{
			EqualizeHelperGetFrequencyTable << <gridDim, blockDim >> > (pPixelsGPU, imageWidth, imageHeight, pFrequencyTableGPU);
		}
		hipMemcpy(pFrequencyTable, pFrequencyTableGPU, sizeof(FrequencyTable), hipMemcpyDeviceToHost);
	}
	hipFree(pPixelsGPU);
	hipFree(pFrequencyTableGPU);
}