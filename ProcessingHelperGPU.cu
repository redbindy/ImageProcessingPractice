#include "hip/hip_runtime.h"
#include "ProcessingHelperGPU.h"

#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>

__global__ void EqualizePixels(ImageDTOForGPU image)
{
	assert(image.pPixels != nullptr);
	assert(image.pFrequencyTable != nullptr);

	const int pixelCount = image.width * image.height;

	const unsigned int col = (blockDim.x * blockIdx.x) + threadIdx.x;
	if (col >= image.width)
	{
		return;
	}

	const unsigned int row = (blockDim.y * blockIdx.y) + threadIdx.y;
	if (row >= image.height)
	{
		return;
	}

	const unsigned int index = row * image.width + col;

	__shared__ float cdf[TABLE_SIZE];

	const int k = threadIdx.y * blockDim.x + threadIdx.x;
	if (k < TABLE_SIZE)
	{
		uint32_t sum = 0;
		for (int i = 0; i <= k; ++i)
		{
			switch (blockIdx.z)
			{
			case EHandleColor::RED:
				{
					sum += image.pFrequencyTable->redTable[i];
				}
				break;

			case EHandleColor::GREEN:
				{
					sum += image.pFrequencyTable->greenTable[i];
				}
				break;

			case EHandleColor::BLUE:
				{
					sum += image.pFrequencyTable->blueTable[i];
				}
				break;

			default:
				printf("Invalid input %d\n", blockIdx.z);
				break;
			}
		}

		cdf[k] = sum / (float)pixelCount;
	}

	__syncthreads();

	Pixel* pPixel = image.pPixels + index;
	switch (blockIdx.z)
	{
	case EHandleColor::RED:
		{
			pPixel->rgba.r = (uint8_t)round(UINT8_MAX * cdf[pPixel->rgba.r]);
		}
		break;

	case EHandleColor::GREEN:
		{
			pPixel->rgba.g = (uint8_t)round(UINT8_MAX * cdf[pPixel->rgba.g]);
		}
		break;

	case EHandleColor::BLUE:
		{
			pPixel->rgba.b = (uint8_t)round(UINT8_MAX * cdf[pPixel->rgba.b]);
		}
		break;

	default:
		assert(false);
		break;
	}
}

__global__ void CalculateFrequencyTable(ImageDTOForGPU image)
{
	assert(image.pPixels != nullptr);
	assert(image.pFrequencyTable != nullptr);

	assert(image.pPixels != nullptr);
	assert(image.pFrequencyTable != nullptr);

	const unsigned int col = (blockDim.x * blockIdx.x) + threadIdx.x;
	if (col >= image.width)
	{
		return;
	}

	const unsigned int row = (blockDim.y * blockIdx.y) + threadIdx.y;
	if (row >= image.height)
	{
		return;
	}

	const unsigned int index = row * image.width + col;

	Pixel* pPixel = image.pPixels + index;
	switch (blockIdx.z)
	{
	case EHandleColor::RED:
		{
			atomicAdd(image.pFrequencyTable->redTable + pPixel->rgba.r, 1);
		}
		break;

	case EHandleColor::GREEN:
		{
			atomicAdd(image.pFrequencyTable->greenTable + pPixel->rgba.g, 1);
		}
		break;

	case EHandleColor::BLUE:
		{
			atomicAdd(image.pFrequencyTable->blueTable + pPixel->rgba.b, 1);
		}
		break;

	default:
		assert(false);
		break;
	}
}

void EqualizeHelperGPU(ImageDTOForGPU image)
{
	assert(image.pPixels != nullptr);
	assert(image.pFrequencyTable != nullptr);

	const int pixelCount = image.width * image.height;

	ImageDTOForGPU imageGPU = { nullptr, image.width, image.height, nullptr };

	hipError_t errorCode = hipMalloc(&(imageGPU.pPixels), sizeof(Pixel) * pixelCount);
	if (errorCode != hipSuccess)
	{
		printf("%s - %s\n", hipGetErrorName(errorCode), hipGetErrorString(errorCode));
		goto CUDA_FREE;
	}

	errorCode = hipMalloc(&(imageGPU.pFrequencyTable), sizeof(FrequencyTable));
	if (errorCode != hipSuccess)
	{
		printf("%s - %s\n", hipGetErrorName(errorCode), hipGetErrorString(errorCode));
		goto CUDA_FREE;
	}

	{
		dim3 blockDim = { 32, 32, 1 };
		dim3 gridDim = {
			(unsigned int)ceil(imageGPU.width / (float)blockDim.x),
			(unsigned int)ceil(imageGPU.height / (float)blockDim.y),
			EHandleColor::COUNT
		};

		hipMemcpy(imageGPU.pPixels, image.pPixels, pixelCount * sizeof(Pixel), hipMemcpyHostToDevice);
		hipMemcpy(imageGPU.pFrequencyTable, image.pFrequencyTable, sizeof(FrequencyTable), hipMemcpyHostToDevice);
		{
			EqualizePixels << <gridDim, blockDim >> > (imageGPU);
		}
		hipMemcpy(image.pPixels, imageGPU.pFrequencyTable, pixelCount * sizeof(Pixel), hipMemcpyDeviceToHost);

		hipMemset(imageGPU.pFrequencyTable, 0, sizeof(FrequencyTable));
		{
			CalculateFrequencyTable << <gridDim, blockDim >> > (imageGPU);
		}
		hipMemcpy(image.pFrequencyTable, imageGPU.pFrequencyTable, sizeof(FrequencyTable), hipMemcpyDeviceToHost);
	}
CUDA_FREE:;
	hipFree(imageGPU.pPixels);
	hipFree(imageGPU.pFrequencyTable);
}

__global__ void Match(ImageDTOForGPU outImage, ImageDTOForGPU srcImage, ImageDTOForGPU refImage)
{
	assert(srcImage.pPixels != nullptr);
	assert(srcImage.pFrequencyTable != nullptr);
	assert(refImage.pPixels != nullptr);
	assert(refImage.pFrequencyTable != nullptr);
	assert(outImage.pPixels != nullptr);
	assert(outImage.pFrequencyTable != nullptr);

	const unsigned int col = (blockDim.x * blockIdx.x) + threadIdx.x;
	if (col >= outImage.width)
	{
		return;
	}

	const unsigned int row = (blockDim.y * blockIdx.y) + threadIdx.y;
	if (row >= outImage.height)
	{
		return;
	}

	const unsigned int index = row * outImage.width + col;

	__shared__ FrequencyTable lookup;

	const int i = threadIdx.y * blockDim.x + threadIdx.x;

	if (i < TABLE_SIZE)
	{
		int k;
		switch (blockIdx.z)
		{
		case EHandleColor::RED:
			{
				for (k = 0; k < TABLE_SIZE; ++k)
				{
					if (refImage.pFrequencyTable->redTable[k] > srcImage.pFrequencyTable->redTable[i])
					{
						break;
					}
				}

				lookup.redTable[i] = k;
			}
			break;

		case EHandleColor::GREEN:
			{
				for (k = 0; k < TABLE_SIZE; ++k)
				{
					if (refImage.pFrequencyTable->greenTable[k] > srcImage.pFrequencyTable->greenTable[i])
					{
						break;
					}
				}

				lookup.greenTable[i] = k;
			}
			break;

		case EHandleColor::BLUE:
			{
				for (k = 0; k < TABLE_SIZE; ++k)
				{
					if (refImage.pFrequencyTable->blueTable[k] > srcImage.pFrequencyTable->blueTable[i])
					{
						break;
					}
				}
				lookup.blueTable[i] = k;
			}
			break;

		default:
			assert(false);
			break;
		}
	}

	__syncthreads();

	Pixel* pOutPixel = outImage.pPixels + index;
	const Pixel srcPixel = srcImage.pPixels[index];
	switch (blockIdx.z)
	{
	case EHandleColor::RED:
		{
			pOutPixel->rgba.r = lookup.redTable[srcPixel.rgba.r];
		}
		break;

	case EHandleColor::GREEN:
		{
			pOutPixel->rgba.g = lookup.greenTable[srcPixel.rgba.g];
		}
		break;

	case EHandleColor::BLUE:
		{
			pOutPixel->rgba.b = lookup.blueTable[srcPixel.rgba.b];
		}
		break;

	default:
		assert(false);
		break;
	}

	pOutPixel->rgba.a = UINT8_MAX;
}

void MatchHelperGPU(ImageDTOForGPU outImage, ImageDTOForGPU srcImage, ImageDTOForGPU refImage)
{
	ImageDTOForGPU srcImageGPU = { nullptr, srcImage.width, srcImage.height, nullptr };
	ImageDTOForGPU refImageGPU = { nullptr, refImage.width, refImage.height, nullptr };
	ImageDTOForGPU outImageGPU = { nullptr, srcImage.width, srcImage.height, nullptr };

	hipError_t errorCode = hipMalloc(&(srcImageGPU.pPixels), sizeof(Pixel) * srcImage.width * srcImage.height);
	if (errorCode != hipSuccess)
	{
		printf("%s - %s\n", hipGetErrorName(errorCode), hipGetErrorString(errorCode));
		goto CUDA_FREE;
	}

	errorCode = hipMalloc(&(srcImageGPU.pFrequencyTable), sizeof(FrequencyTable));
	if (errorCode != hipSuccess)
	{
		printf("%s - %s\n", hipGetErrorName(errorCode), hipGetErrorString(errorCode));
		goto CUDA_FREE;
	}

	errorCode = hipMalloc(&(refImageGPU.pPixels), sizeof(Pixel) * refImage.width * refImage.height);
	if (errorCode != hipSuccess)
	{
		printf("%s - %s\n", hipGetErrorName(errorCode), hipGetErrorString(errorCode));
		goto CUDA_FREE;
	}

	errorCode = hipMalloc(&(refImageGPU.pFrequencyTable), sizeof(FrequencyTable));
	if (errorCode != hipSuccess)
	{
		printf("%s - %s\n", hipGetErrorName(errorCode), hipGetErrorString(errorCode));
		goto CUDA_FREE;
	}

	errorCode = hipMalloc(&(outImageGPU.pPixels), sizeof(Pixel) * outImage.width * outImage.height);
	if (errorCode != hipSuccess)
	{
		printf("%s - %s\n", hipGetErrorName(errorCode), hipGetErrorString(errorCode));
		goto CUDA_FREE;
	}

	errorCode = hipMalloc(&(outImageGPU.pFrequencyTable), sizeof(FrequencyTable));
	if (errorCode != hipSuccess)
	{
		printf("%s - %s\n", hipGetErrorName(errorCode), hipGetErrorString(errorCode));
		goto CUDA_FREE;
	}

	{
		dim3 blockDim = { 32, 32, 1 };
		dim3 gridDim = {
			(unsigned int)ceil(srcImageGPU.width / (float)blockDim.x),
			(unsigned int)ceil(srcImageGPU.height / (float)blockDim.y),
			EHandleColor::COUNT
		};

		hipMemcpy(srcImageGPU.pPixels, srcImage.pPixels, sizeof(Pixel) * srcImage.width * srcImage.height, hipMemcpyHostToDevice);
		hipMemcpy(srcImageGPU.pFrequencyTable, srcImage.pFrequencyTable, sizeof(FrequencyTable), hipMemcpyHostToDevice);

		EqualizePixels << <gridDim, blockDim >> > (srcImageGPU);

		hipMemset(srcImageGPU.pFrequencyTable, 0, sizeof(FrequencyTable));
		CalculateFrequencyTable << <gridDim, blockDim >> > (srcImageGPU);

		hipMemcpy(refImageGPU.pPixels, refImage.pPixels, sizeof(Pixel) * refImage.width * refImage.height, hipMemcpyHostToDevice);
		hipMemcpy(refImageGPU.pFrequencyTable, refImage.pFrequencyTable, sizeof(FrequencyTable), hipMemcpyHostToDevice);

		EqualizePixels << <gridDim, blockDim >> > (refImageGPU);

		hipMemset(refImageGPU.pFrequencyTable, 0, sizeof(FrequencyTable));
		CalculateFrequencyTable << <gridDim, blockDim >> > (refImageGPU);

		Match << <gridDim, blockDim >> > (outImageGPU, srcImageGPU, refImageGPU);

		hipMemset(outImageGPU.pFrequencyTable, 0, sizeof(FrequencyTable));
		CalculateFrequencyTable << <gridDim, blockDim >> > (outImageGPU);

		hipMemcpy(outImage.pPixels, outImageGPU.pPixels, sizeof(Pixel) * outImage.width * outImage.height, hipMemcpyDeviceToHost);
		hipMemcpy(outImage.pFrequencyTable, outImageGPU.pFrequencyTable, sizeof(FrequencyTable), hipMemcpyDeviceToHost);
	}
CUDA_FREE:;
	hipFree(srcImageGPU.pPixels);
	hipFree(srcImageGPU.pFrequencyTable);

	hipFree(refImageGPU.pPixels);
	hipFree(refImageGPU.pFrequencyTable);

	hipFree(outImageGPU.pPixels);
	hipFree(outImageGPU.pFrequencyTable);
}

__global__ void CorrectGamma(ImageDTOForGPU outImage, const float gamma)
{
	assert(outImage.pPixels != nullptr);

	const unsigned int col = (blockDim.x * blockIdx.x) + threadIdx.x;
	if (col >= outImage.width)
	{
		return;
	}

	const unsigned int row = (blockDim.y * blockIdx.y) + threadIdx.y;
	if (row >= outImage.height)
	{
		return;
	}

	constexpr float NORMALIZER = 1 / 255.f;
	constexpr float UNNORMALIZER = 255.f;

	const unsigned int index = row * outImage.width + col;

	Pixel* outPixel = outImage.pPixels + index;
	switch (blockIdx.z)
	{
	case EHandleColor::RED:
		float newR = powf(outPixel->rgba.r * NORMALIZER, gamma);
		newR *= UNNORMALIZER;
		outPixel->rgba.r = static_cast<uint8_t>(roundf(newR));
		break;

	case EHandleColor::GREEN:
		float newG = powf(outPixel->rgba.g * NORMALIZER, gamma);
		newG *= UNNORMALIZER;
		outPixel->rgba.g = static_cast<uint8_t>(roundf(newG));
		break;

	case EHandleColor::BLUE:
		float newB = powf(outPixel->rgba.b * NORMALIZER, gamma);
		newB *= UNNORMALIZER;
		outPixel->rgba.b = static_cast<uint8_t>(roundf(newB));
		break;

	default:
		assert(false);
		break;
	}
}

void GammaHelperGPU(ImageDTOForGPU image, const float gamma)
{
	assert(image.pPixels != nullptr);
	assert(image.pFrequencyTable != nullptr);

	const int pixelCount = image.width * image.height;

	ImageDTOForGPU imageGPU = { nullptr, image.width, image.height, nullptr };

	hipError_t errorCode = hipMalloc(&(imageGPU.pPixels), sizeof(Pixel) * pixelCount);
	if (errorCode != hipSuccess)
	{
		printf("%s - %s\n", hipGetErrorName(errorCode), hipGetErrorString(errorCode));
		goto CUDA_FREE;
	}
	{
		dim3 blockDim = { 32, 32, 1 };
		dim3 gridDim = {
			(unsigned int)ceil(imageGPU.width / (float)blockDim.x),
			(unsigned int)ceil(imageGPU.height / (float)blockDim.y),
			EHandleColor::COUNT
		};

		hipMemcpy(imageGPU.pPixels, image.pPixels, pixelCount * sizeof(Pixel), hipMemcpyHostToDevice);
		{
			CorrectGamma << <gridDim, blockDim >> > (imageGPU, gamma);
		}
		hipMemcpy(image.pPixels, imageGPU.pPixels, pixelCount * sizeof(Pixel), hipMemcpyDeviceToHost);
	}
CUDA_FREE:;
	hipFree(imageGPU.pPixels);
}
