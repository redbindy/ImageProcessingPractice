#include "hip/hip_runtime.h"
#include "ProcessingHelperGPU.h"

#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>

__global__ void EqualizePixels(ImageDTOForGPU imageGPU)
{
	const int pixelCount = imageGPU.width * imageGPU.height;
	
	const unsigned int col = (blockDim.x * blockIdx.x) + threadIdx.x;
	if (col >= imageGPU.width)
	{
		return;
	}

	const unsigned int row = (blockDim.y * blockIdx.y) + threadIdx.y;
	if (row >= imageGPU.height)
	{
		return;
	}

	const unsigned int index = row * imageGPU.width + col;

	__shared__ float cdf[TABLE_SIZE];

	if (threadIdx.x == 0 && threadIdx.y == 0)
	{
		uint32_t sum = 0;
		for (int i = 0; i < TABLE_SIZE; ++i)
		{
			switch (blockIdx.z)
			{
			case EHandleColor::RED:
				{
					sum += imageGPU.pFrequencyTable->redTable[i];
				}
				break;

			case EHandleColor::GREEN:
				{
					sum += imageGPU.pFrequencyTable->greenTable[i];
				}
				break;

			case EHandleColor::BLUE:
				{
					sum += imageGPU.pFrequencyTable->blueTable[i];
				}
				break;

			default:
				printf("Invalid input %d\n", blockIdx.z);
				break;
			}

			cdf[i] = sum / (float)pixelCount;
		}
	}

	__syncthreads();

	Pixel* pPixel = imageGPU.pPixels + index;
	switch (blockIdx.z)
	{
	case EHandleColor::RED:
		{
			pPixel->rgba.r = (uint8_t)round(UINT8_MAX * cdf[pPixel->rgba.r]);
		}
		break;

	case EHandleColor::GREEN:
		{
			pPixel->rgba.g = (uint8_t)round(UINT8_MAX * cdf[pPixel->rgba.g]);
		}
		break;

	case EHandleColor::BLUE:
		{
			pPixel->rgba.b = (uint8_t)round(UINT8_MAX * cdf[pPixel->rgba.b]);
		}
		break;

	default:
		printf("Invalid input %d\n", blockIdx.z);
		break;
	}
}

__global__ void CalculateFrequencyTable(ImageDTOForGPU imageGPU)
{
	const int pixelCount = imageGPU.width * imageGPU.height;

	const unsigned int col = (blockDim.x * blockIdx.x) + threadIdx.x;
	if (col >= imageGPU.width)
	{
		return;
	}

	const unsigned int row = (blockDim.y * blockIdx.y) + threadIdx.y;
	if (row >= imageGPU.height)
	{
		return;
	}

	const unsigned int index = row * imageGPU.width + col;

	Pixel* pPixel = imageGPU.pPixels + index;
	switch (blockIdx.z)
	{
	case EHandleColor::RED:
		{
			atomicAdd(imageGPU.pFrequencyTable->redTable + pPixel->rgba.r, 1);
		}
		break;

	case EHandleColor::GREEN:
		{
			atomicAdd(imageGPU.pFrequencyTable->greenTable + pPixel->rgba.g, 1);
		}
		break;

	case EHandleColor::BLUE:
		{
			atomicAdd(imageGPU.pFrequencyTable->blueTable + pPixel->rgba.b, 1);
		}
		break;

	default:
		printf("Invalid input %d\n", blockIdx.z);
		break;
	}
}

void EqualizeHelperGPU(ImageDTOForGPU image)
{
	const int pixelCount = image.width * image.height;

	ImageDTOForGPU imageGPU = { nullptr, image.width, image.height, nullptr };
	
	hipError_t errorCode = hipMalloc(&(imageGPU.pPixels), sizeof(Pixel) * pixelCount);
	if (errorCode != hipSuccess)
	{
		printf("%s - %s\n", hipGetErrorName(errorCode), hipGetErrorString(errorCode));
		return;
	}

	errorCode = hipMalloc(&(imageGPU.pFrequencyTable), sizeof(FrequencyTable));
	if (errorCode != hipSuccess)
	{
		printf("%s - %s\n", hipGetErrorName(errorCode), hipGetErrorString(errorCode));
		return;
	}

	{
		dim3 blockDim = { 32, 32, 1 };
		dim3 gridDim = {
			(unsigned int)ceil(imageGPU.width / (float)blockDim.x),
			(unsigned int)ceil(imageGPU.height / (float)blockDim.y),
			EHandleColor::COUNT
		};

		hipMemcpy(imageGPU.pPixels, image.pPixels, pixelCount * sizeof(Pixel), hipMemcpyHostToDevice);
		hipMemcpy(imageGPU.pFrequencyTable, image.pFrequencyTable, sizeof(FrequencyTable), hipMemcpyHostToDevice);
		{
			EqualizePixels << <gridDim, blockDim >> > (imageGPU);
		}
		hipMemcpy(image.pPixels, imageGPU.pFrequencyTable, pixelCount * sizeof(Pixel), hipMemcpyDeviceToHost);

		hipMemset(imageGPU.pFrequencyTable, 0, sizeof(FrequencyTable));
		{
			CalculateFrequencyTable << <gridDim, blockDim >> > (imageGPU);
		}
		hipMemcpy(image.pFrequencyTable, imageGPU.pFrequencyTable, sizeof(FrequencyTable), hipMemcpyDeviceToHost);
	}
	hipFree(imageGPU.pPixels);
	hipFree(imageGPU.pFrequencyTable);
}