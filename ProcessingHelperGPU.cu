#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include "ProcessingHelperGPU.h"

#include <stdio.h>


static __global__ void Test()
{
	printf("Test");
}

void CallTest()
{
	Test<<<1, 10, 10 >>>();
}
